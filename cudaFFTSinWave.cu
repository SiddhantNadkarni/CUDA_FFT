#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <math.h>
#include <fstream>
#include <chrono>


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>


typedef float2 Complex;

#define NX 1024
#define BATCH 1 


int main()
{
    hipError_t err = hipSuccess;
    float mulApl = 1.0;
    float mulFre = 2*M_PI;
    float divFreq = 20.0;

    hipfftHandle plan;
    hipfftComplex *data = nullptr; //hipfftComplex is single-precision, floating-point Complex

    //allocate GPU memory
    if(hipMalloc((void **)&data, sizeof(hipfftComplex)*NX*BATCH) != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate memory for data, error = %s \n ", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //create signal on host
    Complex* h_signal = new Complex [NX*BATCH];

    for(int i = 0; i < NX; i++)
    {
        h_signal[i].x = mulApl*sin(mulFre*i / divFreq);
        h_signal[i].y = 0;
    }

    std::ofstream input_file;
    std::ofstream output_file;

    input_file.open("input_file.dat");
    output_file.open("output_file.dat");


    for(int i = 0; i < NX; i++)
    {
        input_file << h_signal[i].x << '\n';
    }

    
    

    //allocate memory on GPU
    if(hipMemcpy(data, h_signal, sizeof(Complex)*NX*BATCH, hipMemcpyHostToDevice)!=hipSuccess)
    {
        fprintf(stderr, "CuFFT error: hipMemcpy host to device failed! \n");
        exit(EXIT_FAILURE);
    }

    

    //create a plan for 1D transform
    if(hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS) 
    {
        fprintf(stderr, "CuFFT error: hipfftPlan1d failed! \n");
        exit(EXIT_FAILURE);
    }

    auto start = std::chrono::system_clock::now();

    //complex-to-complex transforms for single/double precision
    if(hipfftExecC2C(plan, data, data, HIPFFT_FORWARD) != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "CuFFT error: hipfftExecC2C failed! \n");
        exit(EXIT_FAILURE);
    }

    auto finish = std::chrono::system_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start);

    std::cout << "time taken for FFT calculation of NX samples: " << duration.count() << " nanoseconds." << '\n';

    //Synchronize device i.e. barrier
    if(hipDeviceSynchronize() != hipSuccess)
    {
        fprintf(stderr, "Device failed to Synchronize! error = %s \n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    Complex* h_convolved_signal = h_signal;
    for(int i =0; i< NX; i++)
    {
        h_convolved_signal[i].x = 0;
        h_convolved_signal[i].y = 0;
    }

    if(hipMemcpy(h_convolved_signal, data, sizeof(Complex)*NX*BATCH, hipMemcpyDeviceToHost) != hipSuccess)
    {
        fprintf(stderr, "Device to host data copy failed, error = %s \n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    for(int i = 0; i < NX; i++)
    {
        // std::cout << i << ". cos: " <<  h_convolved_signal[i].x << ", sin: " << h_convolved_signal[i].y << '\n';
        double val = pow(pow(h_convolved_signal[i].x, 2) + pow(h_convolved_signal[i].y, 2), 0.5);
        output_file << val << '\n';
    }

    //clear FFT allocated resources
    if(hipfftDestroy(plan)!= HIPFFT_SUCCESS)
    {
        fprintf(stderr, "Failed to free plan data resources");
        exit(EXIT_FAILURE);
    }

    //clear GPU memory
    if(hipFree(data)!= hipSuccess)
    {
        fprintf(stderr, "Failed to free data memory, error = %s \n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //reset GPU
    if(hipDeviceReset() != hipSuccess)
    {
        fprintf(stderr, "Failed to reset device, error = %s", hipGetErrorString(err));
    }

    //clear Host memory
    delete [] h_signal;

    output_file.close();
    input_file.close();

    
    return 0; 
}